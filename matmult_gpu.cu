
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#define VEC_SIZE 10
#define BLOCK_SIZE 16

double gettimeofday_sec();

__global__ void VecAddKernel(float *, float *, float *);

int main(){
  double t1, t2;
  float *A, *B, *C;
  hipHostMalloc((void**)&A ,sizeof(float)*VEC_SIZE, hipHostMallocDefault);
  hipHostMalloc((void**)&B ,sizeof(float)*VEC_SIZE, hipHostMallocDefault);
  hipHostMalloc((void**)&C ,sizeof(float)*VEC_SIZE, hipHostMallocDefault);

  float *d_A, *d_B, *d_C;
  hipMalloc((void**)&d_A, sizeof(float)*VEC_SIZE);
  hipMalloc((void**)&d_B, sizeof(float)*VEC_SIZE);
  hipMalloc((void**)&d_C, sizeof(float)*VEC_SIZE);

  hipMemcpy(d_A, A, sizeof(float)*VEC_SIZE, hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, sizeof(float)*VEC_SIZE, hipMemcpyHostToDevice);

  dim3 dimBlock(BLOCK_SIZE, 1);
  dim3 dimGrid(VEC_SIZE/BLOCK_SIZE, 1);
  VecAddKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
  
  t1 = gettimeofday_sec();
  hipMemcpy(C, d_C, sizeof(float)*VEC_SIZE, hipMemcpyDeviceToHost);
  t2 = gettimeofday_sec();
  
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);

  hipHostFree(A);
  hipHostFree(B);
  hipHostFree(C);
  printf("Run Time: %f[s]", t2 - t1);
}

__global__ void VecAddKernel(float *d_A, float *d_B, float *d_C) {
  
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  d_C[tid] = d_A[tid] + d_B[tid];
}

double gettimeofday_sec(){
  struct timeval tv;
  gettimeofday(&tv, NULL);
  return tv.tv_sec + (double)tv.tv_usec*1e-6;
}
