#include "hip/hip_runtime.h"
#define MAT_SIZE 32 

__global__ void MatrixMult(float *, float *, float *);

int main(){
  float *A, *B, *C;
  hipHostMalloc((void**)&A ,sizeof(float)*MAT_SIZE*MAT_SIZE);
  hipHostMalloc((void**)&B ,sizeof(float)*MAT_SIZE*MAT_SIZE);
  hipHostMalloc((void**)&C ,sizeof(float)*MAT_SIZE*MAT_SIZE);

  float *d_A, *d_B, *d_C;
  hipMalloc((void**)&d_A, sizeof(float)*MAT_SIZE*MAT_SIZE);
  hipMalloc((void**)&d_B, sizeof(float)*MAT_SIZE*MAT_SIZE);
  hipMalloc((void**)&d_C, sizeof(float)*MAT_SIZE*MAT_SIZE);

  hipMemcpy(d_A, A, sizeof(float)*MAT_SIZE*MAT_SIZE, hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, sizeof(float)*MAT_SIZE*MAT_SIZE, hipMemcpyHostToDevice);

  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
  dim3 dimGrid(MAT_SIZE/4, MAT_SIZE/4);
  MatrixMult<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);

  hipMemcpy(C, d_C, sizeof(float)*MAT_SIZE*MAT_SIZE, hipMemcpyDeviceToHost);

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);

  hipHostFree(A);
  hipHostFree(B);
  hipHostFree(C);
}

__global__ void MatrixMult(float *d_A, float *d_B, float *d_C) {
  int i;

  unsigned int row_idx = blockDim.x * blockIdx.x + threadIdx.x;
  unsigned int col_idx = blockDim.y * blockIdx.y * threadIdx.y;


  for(i = 0, i < MAT_SIZE; i++) {
     

}
