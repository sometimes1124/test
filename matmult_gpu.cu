#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "gettimeofday_sec.c"
#define MAT_SIZE 32
#define BLOCK_SIZE 16

__global__ void MatMultKernel(float *, float *, float *);

int main(){
  int i;
  int size = sizeof(float)*MAT_SIZE*MAT_SIZE;
  double t1, t2;
  float *A, *B, *C;
  hipHostMalloc((void**)&A ,size);
  hipHostMalloc((void**)&B ,size);
  hipHostMalloc((void**)&C ,size);
  
  srand((unsigned) time(NULL));
  for(i = 0; i < MAT_SIZE * MAT_SIZE; i++){
    A[i] = rand();
    B[i] = rand();
  }
  
  float *d_A, *d_B, *d_C;
  hipMalloc((void**)&d_A, size);
  hipMalloc((void**)&d_B, size);
  hipMalloc((void**)&d_C, size);

  hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
  dim3 dimGrid(MAT_SIZE/BLOCK_SIZE, MAT_SIZE/BLOCK_SIZE);
  t1 = gettimeofday_sec();
  MatMultKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
  t2 = gettimeofday_sec();

  hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);

  hipHostFree(A);
  hipHostFree(B);
  hipHostFree(C);
  printf("Run Time: %f[s]", t2 - t1);
}

__global__ void MatMultKernel(float *d_A, float *d_B, float *d_C) {
  int i;
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int idy = blockIdx.y * blockDim.y + threadIdx.y;

  d_C[idy*MAT_SIZE+idy] = 0.0;
  for(i = 0; i < MAT_SIZE; i++) {
    d_C[idy*MAT_SIZE+idx] = d_A[idy*MAT_SIZE+i] * d_B[i*MAT_SIZE+idx];
  }
}
