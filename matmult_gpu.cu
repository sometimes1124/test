#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>
#include "gettimeofday_sec.c"

__global__ void MatMultKernel(float *, float *, int , float *);


int main(int argc, char *argv[]){
  int MAT_SIZE = atoi(argv[1]);
  int iter = atoi(argv[2]);
  int BLOCK_SIZE = atoi(argv[3]);
  int i;
  int size = sizeof(float)*MAT_SIZE*MAT_SIZE;
  double t1, t2;
  float *A, *B, *C;
  hipHostMalloc((void**)&A ,size);
  hipHostMalloc((void**)&B ,size);
  hipHostMalloc((void**)&C ,size);
  
  srand((unsigned) time(NULL));
  for(i = 0; i < MAT_SIZE * MAT_SIZE; i++){
    A[i] = rand();
    B[i] = rand();
  }
  
  float *d_A, *d_B, *d_C;
  hipMalloc((void**)&d_A, size);
  hipMalloc((void**)&d_B, size);
  hipMalloc((void**)&d_C, size);

  hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
  dim3 dimGrid(MAT_SIZE/BLOCK_SIZE, MAT_SIZE/BLOCK_SIZE);
  t1 = gettimeofday_sec();
  for(i = 0; i < iter; i++){
    MatMultKernel<<<dimGrid, dimBlock>>>(d_A, d_B, MAT_SIZE, d_C);
  }
  t2 = gettimeofday_sec();

  hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);

  hipHostFree(A);
  hipHostFree(B);
  hipHostFree(C);
  printf("Run Time: %.3le[s]", (t2 - t1)/iter);
}

__global__ void MatMultKernel(float *d_A, float *d_B, int MAT_SIZE,float *d_C) {
  int i;
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int idy = blockIdx.y * blockDim.y + threadIdx.y;

  d_C[idy*MAT_SIZE+idy] = 0.0;
  for(i = 0; i < MAT_SIZE; i++) {
    d_C[idy*MAT_SIZE+idx] = d_A[idy*MAT_SIZE+i] * d_B[i*MAT_SIZE+idx];
  }
}
