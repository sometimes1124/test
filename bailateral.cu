#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "gettimeofday_sec.c"
#define  WIDTH 1024
#define  HEIGHT 768
#define  BLOCK_SIZE 32

__global__ void bailateral(int*, int*);

int main() { 
  double t1, t2;
  int i;
  int size = sizeof(int)*WIDTH*HEIGHT;
  FILE *fp;

  int *d_im, *d_out;

  //hipHostMalloc((void**)&im,  size);
  //hipHostMalloc((void**)&out, size);
  int *im = (int*)malloc(size);
  int *out = (int*)malloc(size);


  fp = fopen("photo.dat", "r");
  for(i = 0; i < HEIGHT * WIDTH; i++) {
    fscanf(fp, "%d", &im[i]);
    }
  fclose(fp);

  hipMalloc((void**)&d_im,  size);
  hipMalloc((void**)&d_out, size);

  hipMemcpy(d_im, im, size, hipMemcpyHostToDevice);

  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
  dim3 dimGrid(WIDTH/BLOCK_SIZE, HEIGHT/BLOCK_SIZE);
  t1 = gettimeofday_sec();
  bailateral<<<dimGrid, dimBlock>>>(d_im, d_out);
  t2 = gettimeofday_sec();

  hipMemcpy(out, d_out, size, hipMemcpyDeviceToHost);
  hipFree(d_im);
  hipFree(d_out);

  fp = fopen("out.dat", "w");
  for(i = 0; i < HEIGHT*WIDTH; i++) {
    fprintf(fp, "%d\n", out[i]);
    }
  fclose(fp);

  free(im);
  free(out);
  
  printf("Run Time: %f[s]", t2 - t1);

  return 0;
}


__global__ void bailateral(int *d_im, int *d_out) {
  
  int m, n;
  
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  int j = blockIdx.y*blockDim.y + threadIdx.y;

  const int w = 3;
  const int sigma1 = 30;
  const int sigma2 = 30;

  float num = 0;
  float denom = 0;
  for(m = -w; m <= w; m++) {
    for(n = -w; n <= w; n++) {
      if(j + n < 0 || j + n >= HEIGHT || 
         i + m < 0 || i + m >= WIDTH)
        continue;
      float t = d_im[WIDTH*j + i] - d_im[WIDTH*(j + n) + (i + m)];
      float s = expf(-(m * m + n * n) / (2 * sigma1 * sigma1)) * 
        expf(-(t * t) / (2 * sigma2 * sigma2));
      num += d_im[WIDTH*(j + n) + (i + m)] * s;
      denom += s;
    }
  }
  if(denom == 0)
    denom = 1;
  d_out[WIDTH*j+i] = num / denom;
 }





