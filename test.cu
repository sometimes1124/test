
#include <hip/hip_runtime.h>
#include<iostream>
#include<vector>

__global__
void vecadd(float *a, float *b, float *c, int num)
{
  c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}


int main(int argc, char *argv[])
{
  const int num = 16;
  std::vector<float> a(num, 1);
  std::vector<float> b(num, 1);
  std::vector<float> c(num, 0);
  
  float *d_a;
  float *d_b;
  float *d_c;
  
  hipMalloc(&d_a, num * sizeof(float));
  hipMalloc(&d_b, num * sizeof(float));
  hipMalloc(&d_c, num * sizeof(float));

  hipMemcpy(d_a, &a[0], num*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_a, &b[0], num*sizeof(float), hipMemcpyHostToDevice);
  
  dim3 grid_size = dim3(1, 1, 1);
  dim3 block_size = dim3(num, 1, 1);
  
  vecadd<<<grid_size, block_size>>>(d_a, d_b, d_c, num);
 
  hipMemcpy(&c[0], d_c, num*sizeof(float), hipMemcpyDeviceToHost);
  
  for(int i=0; i < num; ++i) std::cout << c[i]  << std::endl;
  
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
  
  

  return 0;
}
