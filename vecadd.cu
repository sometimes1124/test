#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#define VEC_SIZE 8
#define BLOCK_SIZE 4

__global__ void VecAddKernel(float *, float *, float *);

int main(){
  int i;
  float *A, *B, *C;
  hipHostMalloc((void**)&A ,sizeof(float)*VEC_SIZE);
  hipHostMalloc((void**)&B ,sizeof(float)*VEC_SIZE);
  hipHostMalloc((void**)&C ,sizeof(float)*VEC_SIZE);

  srand((unsigned int)time(NULL));
  for(i = 0; i < VEC_SIZE; i++) {
    A[i] = rand();
    B[i] = rand();
  }

  float *d_A, *d_B, *d_C;
  hipMalloc((void**)&d_A, sizeof(float)*VEC_SIZE);
  hipMalloc((void**)&d_B, sizeof(float)*VEC_SIZE);
  hipMalloc((void**)&d_C, sizeof(float)*VEC_SIZE);

  hipMemcpy(d_A, A, sizeof(float)*VEC_SIZE, hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, sizeof(float)*VEC_SIZE, hipMemcpyHostToDevice);

  dim3 dimBlock(BLOCK_SIZE, 1);
  dim3 dimGrid(VEC_SIZE/BLOCK_SIZE, 1);
  VecAddKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);

  hipMemcpy(C, d_C, sizeof(float)*VEC_SIZE, hipMemcpyDeviceToHost);
  
  for(i = 0; i < VEC_SIZE; i++){
    printf("%.0f + %.0f = %.0f\n", A[i], B[i], C[i]);
  }

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);

  hipHostFree(A);
  hipHostFree(B);
  hipHostFree(C);
}

__global__ void VecAddKernel(float *d_A, float *d_B, float *d_C) {
  
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  d_C[tid] = d_A[tid] + d_B[tid];
}
