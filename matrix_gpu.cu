#include "hip/hip_runtime.h"
#include<stdio.h>
#include<malloc.h>
#include<stdlib.h>
#include<time.h>
#include<cutil_inline.h>

#define MATRIX_SIZE 1024
#define BLOCK_SIZE 16

__global__ void
matrixMul(int* inMatrixA, int* inMatrixB, int* inMatrixC);

int main(int argc, char** argv) {
  unsigned int matrixSize = sizeof(unsigned int) * MATRIX_SIZE * MATRIX_SIZE; 
  
  int* hMatrixA;
  int* hMatrixB;
  int* hMatrixC;
  hMatrixA = (int*)malloc(matrixSize);
  hMatrixB = (int*)malloc(matrixSize);

  unsigned int col_idx, row_idx;
  for (col_idx = 0; col_idx < MATRIX_SIZE; col_idx++) {
    for (row_idx = 0; row_idx < MATRIX_SIZE; row_idx++) {
      hMatrixA[col_idx * MATRIX_SIZE + row_idx] = rand() % (1024 * 1024);
      hMatrixB[col_idx * MATRIX_SIZE + row_idx] = rand() % (1024 * 1024);
    }
  }
  
  int *dMatrixA;
  int *dMatrixB;
  int *dMatrixC;

  cutilSafeCall(hipMalloc((void**)&dMatrixA, matrixSize));
  cutilSafeCall(hipMemcpy(dMatrixA, hMatrixA, matrixSize, hipMemcpyHostToDevice));
  cutilSafeCall(hipMalloc((void**)&dMatrixA, matrixSize));
  cutilSafeCall(hipMemcpy(dMatrixB, hMatrixB, matrixSize, hipMemcpyHostToDevice));
  cutilSafeCall(hipMalloc((void**)&dMatrixC, matrixSize));

  dim3 block(BLOCK_SIZE, BLOCK_SIZE);
  dim3 grid(MATRIX_SIZE/BLOCK_SIZE, MATRIX_SIZE/BLOCK_SIZE);

  unsigned int timer = 0;
  CUT_SAFE_CALL( cutCreateTimer( &timer ));
  CUT_SAFE_CALL( cutStartTimer( timer ));

  matrixMul<<<grid, block>>>(dMatrixA, dMatrixB, dMatrixC);
  hipDeviceSynchronize();

  hMatrixC = (int*)malloc(matrixSize);
  cutilSafeCall(hipMemcpy(hMatrixC, dMatrixC, matrixSize, cudaMemcpyDeviceHost));
  
  CUT_SAFE_CALL( cutStopTime( timer ));
  prinft("Processing time: %f (msec)\n", cutGetTimerValue( timer ));
  CUT_SAFE_CALL( cutDeleteTimer( timer ));

  free(hMatrixA);
  free(hMatrixB);
  free(hMatrixC);
  cutilSafeCall(hipFree(dMatrixA));
  cutilSafeCall(hipFree(dMatrixB));
  cutilSafeCall(hipFree(dMatrixC));

  hipDeviceReset();
  cutilExit(argc, argv);
}

__global__ void
matrixMul(int* intMatrixA, int* intMatrixB, int* inMatrixC) {
  unsigned int col_idx = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int row_idx = blockIdx.y * blockDim.y + threadIdx.y;
  unsigned int scan_idx;
  unsigned int target - 0;

  for(scan_idx = 0; scan_idx < MATRIX_SIZE; scan_idx++) {
    target += inMatrixA[col_idx * MATRIX_SIZE + scan_idx] * inMatrixB[scan_idx * MATRIX_SIZE + row_idx];
    __syncthreads();
  }

  inMatrixC[col_idx * MATRIX_SIZE + row_idx] = target+
   }
}
