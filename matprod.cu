#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<sys/time.h>
#include"util.c"

//プロトタイプ宣言
__global__ void matrixProductKernel(float *, float *, float *, int);
void matrixProduct(float *, float *, float *, int);

/**
   n次正方行列の積を計算
   CPU実装とGPU実装それぞれの実行時間を測定

   引数(デフォルト)
   [1]: 正方行列の次数(32)
   [2]: ブロックの次数(16)
   [3]: 各方式の試行回数(3)
 */
int main(int argc, char **argv){
  int i;

  //パラメータ設定
  int n = 32;         //正方行列の次数
  int blockSize = 16; //ブロックの次数
  int n_trial = 3;    //試行回数
  switch(argc){
  case 4: sscanf(argv[3], "%d", &n_trial);
  case 3: sscanf(argv[2], "%d", &blockSize);
  case 2: sscanf(argv[1], "%d", &n);
  case 1: break;
  default:
    fprintf(stderr, "usage:\n[1]n (32)\n[2]block_size (16)\n[3]n_trial (3)\n");
    exit(1);
  }
  int size = n*n;

  //ホストメモリの確保
  float *A = (float *)malloc(sizeof(float) * size); 
  float *B = (float *)malloc(sizeof(float) * size); 
  float *C = (float *)malloc(sizeof(float) * size); 

  //行列データ格納(乱数)
  srand((unsigned) time(NULL));
  for(i = 0; i < n*n; i++){
    A[i] = rand();
    B[i] = rand();
  }

  //デバイスメモリの確保
  float *d_A, *d_B, *d_C;
  hipMalloc((void **)&d_A, sizeof(float) * size);
  hipMalloc((void **)&d_B, sizeof(float) * size);
  hipMalloc((void **)&d_C, sizeof(float) * size);
  
  //ホストからデバイスへデータ転送
  hipMemcpy(d_A, A, sizeof(float) * size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, sizeof(float) * size, hipMemcpyHostToDevice);

  //CPU実装の実行時間測定
  double tmp = 0.0;
  double c_tb, c_te;
  printf("(n: %d, bsize: %d, trial: %d)\n", n, blockSize, n_trial);
  for(i = 0; i < n_trial; i++){
    c_tb = gettimeofday_sec();
    matrixProduct(A, B, C, n);
    c_te = gettimeofday_sec();
    tmp += c_te - c_tb;
    printf(".");
  }
  printf("CPU_time = %.3le\n", tmp/n_trial);

  //GPU実装の実行時間測定
  dim3 dimBlock(blockSize, blockSize);
  dim3 dimGrid(n/blockSize, n/blockSize);
  double g_tb, g_te;
  tmp = 0.0;
  for(i = 0; i < n_trial; i++){
    g_tb = gettimeofday_sec();
    matrixProductKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, n);
    hipDeviceSynchronize();
    g_te = gettimeofday_sec();
    tmp += g_te - g_tb;
    printf(".");
  }
  printf("GPU_time = %.3le\n", tmp/n_trial);

  //デバイスからホストへデータ転送
  hipMemcpy(C, d_C, sizeof(float) * size, hipMemcpyDeviceToHost);

  //メモリ解放
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  free(A);
  free(B);
  free(C);
}

//正方行列の積(GPU)
__global__ void matrixProductKernel(float *d_A, float *d_B, float *d_C, int n){
  int i;

  int tidx = blockIdx.x * blockDim.x + threadIdx.x;
  int tidy = blockIdx.y * blockDim.y + threadIdx.y;

  d_C[tidy*n+tidx] = 0.0;
  for(i = 0; i < n; i++){
      d_C[tidy*n+tidx] += d_A[tidy*n+i] * d_B[i*n+tidx];
  }
 
}

//正方行列の積(CPU)
void matrixProduct(float *d_A, float *d_B, float *d_C, int n){
  int i, x, y, yn, ynx;

  for(y = 0; y < n; y++){
    yn = y*n;
    for(x = 0; x < n; x++){
      ynx = yn + x;
      d_C[ynx] = 0.0;
      for(i = 0; i < n; i++){
        d_C[ynx] += d_A[yn+i] * d_B[i*n+x];
      }
    }
  }

}
